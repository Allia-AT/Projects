#include "hip/hip_runtime.h"
#include "com2039.hpp"


size_t loadSamples(string fileName, float **addressToSamples){

	string line;
	std::ifstream inputFile(fileName);
	// Count the number of lines to allocate
	// the right amount of memory
	size_t numLines=0;
	while( getline( inputFile, line ) ){
		numLines++;
	};

	// Allocate memory on CPU
	// Notice that we dereference adressToSamples
	*addressToSamples  = (float*)malloc(numLines*sizeof(float));

	// Return to the beginning of the file
	// and fill in the
	inputFile.clear();
	inputFile.seekg(0);
	for(int j = 0; j < numLines; j++){
		getline(inputFile, line);
		(*addressToSamples)[j] = std::stof(line);
	}

	inputFile.close();
	return numLines;
}

/////// Find Maximum
__global__ void maxReduceKernel(float *d_in, int lenArray){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x;
		float val = 0;
		float arr[BLOCK_SIZE];
		//find max value in each thread
		for(int i = index; i < lenArray; i += stride){
			d_in[index] = fmaxf(d_in[index], d_in[i]);
		}

		//printf("%f\n", d_in[index]);
		__syncthreads();

		//block_size = number of threads in a block
			//so put thread max into shared memory for each block
		__shared__ float threadSharedMem[BLOCK_SIZE];

		threadSharedMem[threadIdx.x] = d_in[index];
		__syncthreads();
		//make shared memory

		__shared__ float sharedMem[BLOCK_SIZE];

		for (int i=0; i < BLOCK_SIZE; ++i){
			//itterate through threadShareMem and compare to
			//sharedMem[block.id] and put the min value into
			//it
			val = fmaxf(val, threadSharedMem[i]);
		}
		__syncthreads();

		sharedMem[blockIdx.x] = val;
		__syncthreads();

		//now itterate though this array to find the min
		//out of all the blocks

		for (int i=0; i < BLOCK_SIZE; ++i){
			val = fmaxf(val, sharedMem[i]);
		}
		__syncthreads();

		//write back to global
		d_in[0] = val;

}


float findMaxValue(float* samples_h, size_t numSamples){
	//allocate memory
	int *device_array = 0;
	int *host_array = 0;
	float *tempMax;
	float maxVal = 0;

	host_array = (int*)malloc(BLOCK_SIZE * sizeof(int));
	hipMalloc((void**)&device_array, BLOCK_SIZE);
	hipMallocManaged(&tempMax, numSamples*sizeof(float));

	//send data to the GPU from the CPU
	for (int i=0; i < numSamples; i++){
		tempMax[i] = samples_h[i];
	}
	hipMemcpy(device_array, tempMax, numSamples*sizeof(float), hipMemcpyHostToDevice);

	//make the threads and put them to work
	maxReduceKernel<<<ceil(numSamples/BLOCK_SIZE), BLOCK_SIZE>>>(tempMax, numSamples);
	hipError_t err = hipDeviceSynchronize();
//	printf("Run kernel: %s\n", hipGetErrorString(err));
//	err = hipDeviceSynchronize();
//	printf("Run kernel: %s\n", hipGetErrorString(err));

	//return a copy of [0] element (this should be the min value)
	maxVal = tempMax[0];
	hipFree(tempMax);
	hipFree(device_array);
	hipFree(host_array);
	return maxVal;
}


/////// Find Minimum
__global__ void minReduceKernel(float *d_in, int lenArray){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x;
	float val = 0;
	float arr[BLOCK_SIZE];
	//find min value in each thread
	for(int i = index; i < lenArray; i += stride){
		d_in[index] = fminf(d_in[index], d_in[i]);
	}

	//printf("%f\n", d_in[index]);
	__syncthreads();

	//block_size = number of threads in a block
		//so put thread max into shared memory for each block
	__shared__ float threadSharedMem[BLOCK_SIZE];

	threadSharedMem[threadIdx.x] = d_in[index];
	__syncthreads();
	//make shared memory

	__shared__ float sharedMem[BLOCK_SIZE];

	for (int i=0; i < BLOCK_SIZE; ++i){
		//itterate through threadShareMem and compare to
		//sharedMem[block.id] and put the min value into
		//it
		val = fminf(val, threadSharedMem[i]);
	}
	__syncthreads();

	sharedMem[blockIdx.x] = val;
	__syncthreads();

	//now itterate though this array to find the min
	//out of all the blocks

	for (int i=0; i < BLOCK_SIZE; ++i){
		val = fminf(val, sharedMem[i]);
	}
	__syncthreads();

	//write back to global
	d_in[0] = val;

}



float findMinValue(float* samples_h, size_t numSamples){
	//allocate memory
	int *device_array = 0;
	int *host_array = 0;
	float *temp;
	float minVal = 0;

	host_array = (int*)malloc(BLOCK_SIZE * sizeof(int));
	hipMalloc((void**)&device_array, BLOCK_SIZE);
	hipMallocManaged(&temp, numSamples*sizeof(float));

	//send data to the GPU from the CPU
	for (int i=0; i < numSamples; i++){
		temp[i] = samples_h[i];
	}
	hipMemcpy(device_array, temp, numSamples*sizeof(float), hipMemcpyHostToDevice);

	//make the threads and put them to work
	minReduceKernel<<<ceil(numSamples/BLOCK_SIZE), BLOCK_SIZE>>>(temp, numSamples);

	hipError_t err = hipDeviceSynchronize();
//	printf("Run kernel: %s\n", hipGetErrorString(err));
//	err = hipDeviceSynchronize();
//	printf("Run kernel: %s\n", hipGetErrorString(err));

	//return a copy of [0] element (this should be the min value)
	minVal = temp[0];
	hipMemcpy(device_array, temp, numSamples*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(temp);
	return minVal;
}



/////// Create Histogram
__global__ void histogramKernel256(float *d_in, unsigned int *hist, size_t lenArray, float minValue, float maxValue) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x;
	int binIndex;
	int Cval = 0;
	bool flag = false;
	//want hist with 256 bins with equal width.

	float binWidth = (maxValue-minValue) /NUM_BINS;
	if (index == 0){
		printf("bin width: %f\n", binWidth);
	};


	__shared__ int frequencySharedMem[NUM_BINS];
	for (int i = 0; i < NUM_BINS; i++){
		frequencySharedMem[i] = 0;
	}

	__syncthreads();

	for (int i=index; i < lenArray; i += stride){
		binIndex = (d_in[i] - minValue) / binWidth;
		if (binIndex > 255){
			binIndex = 255;
		}
//		if (threadIdx.x ==0){
//			printf("\nBin [%i]", binIndex);
//		}
		__syncthreads();
		atomicAdd(&frequencySharedMem[binIndex], 1);
		__syncthreads();
//		if (threadIdx.x ==0){
//			printf(" | %i", frequencySharedMem[binIndex]);
//		}
//		__syncthreads();
	}
//	__syncthreads();
	if (index ==0){
		Cval = 0;
//		for (int i = 0; i < NUM_BINS; i++){
//			printf("\nBin : %i | %f", i, frequencySharedMem[i]);
//			Cval += frequencySharedMem[i];
//		}
		printf("\ntotal frequency: %i", Cval);
	};

	//------------------------------------assuming frequency found-----------------------------------------------------

	__shared__ float frequencyDensitySharedMem[NUM_BINS];
	if (index == 0){
		int counter = 0;
		for (int i = 0; i < NUM_BINS; i++){
			frequencyDensitySharedMem[i] = (frequencySharedMem[i] / binWidth);
			printf("\nBin[%i]: %f", i, frequencyDensitySharedMem[i]);
			counter += frequencySharedMem[i];
		}
		printf("\nTotal number of elements in histogram: %i", counter);

	}
	__syncthreads();

	for (int i = 0; i < NUM_BINS; i++){
		hist[i] = frequencyDensitySharedMem[i];
	}

}

//for (int i=index; i < lenArray; i += stride){
//		cumulativeBinWidth = (d_in[index] / binWidth) + 1;
//		int currentBin = cumulativeBinWidth / binWidth;
//		__syncthreads();
//		atomicAdd(&frequencySharedMem[currentBin], 1);
//
//		__syncthreads();
//	}

//while (index < lenArray) {
//		//atomicAdd(int* address, int val)
//			//When a thread executes this operation,
//			//a memory address is read, has the value of
//			//‘val’ added to it, and the result is
//			//written back to memory.
//		//atomicAdd( &(hist[d_in[index]]), 1);
//		index += stride;
//	}
//	__syncthreads();

//Cval = frequencySharedMem[currentBin];
//			__syncthreads();
//
//			Cval += 1;
//			__syncthreads();
//			frequencySharedMem[currentBin] = Cval;


/// histogram
void histogram256(float *samples_h, size_t numSamples, unsigned int **hist_h, float minValue, float maxValue) {

	int *device_array = 0;
	int *host_array = 0;
	float *buffer;
	unsigned int *hist = *hist_h;
	int grid_size = (numSamples + BLOCK_SIZE -1)/BLOCK_SIZE;
	printf("num blocks: %d\n", grid_size);

	host_array = (int*)malloc(BLOCK_SIZE * sizeof(int));
	hipMalloc((void**)&device_array, BLOCK_SIZE);
	hipMallocManaged(&buffer, numSamples*sizeof(float));
	hipMemset(hist, 0, NUM_BINS * sizeof(float));
//	for (int i=0; i <NUM_BINS; i++){
//		hist[i] = *hist_h[i];
//	}

	for (int i = 0; i < numSamples; i++) {
		buffer[i] = samples_h[i];
	}

	hipMemcpy(device_array, buffer, numSamples*sizeof(float), hipMemcpyHostToDevice);


	histogramKernel256<<<grid_size, BLOCK_SIZE>>>(buffer, hist, numSamples, minValue, maxValue);

	hipDeviceSynchronize();
	for (int i=0; i < 5; i++){
			printf("%f", hist[i]);
		}
	hipGetLastError();
	hipMemcpy(device_array, buffer, numSamples*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(buffer);
}

